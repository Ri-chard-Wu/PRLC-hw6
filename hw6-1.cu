
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <cstring>

#define N 1024
typedef unsigned int WORD;
typedef unsigned char BYTE;
using namespace std;

// double min(double a, double b) {return a < b ? a : b;}

double CPU_reduction(double *arr, int n) {

    double ret = arr[0];

    for (int i = 1; i < n; i++) {
        ret = min(ret, arr[i]);
    }

    return ret;
}



double fRand(double fMin, double fMax)
{
    double f = (double)rand() / RAND_MAX;
    return fMin + f * (fMax - fMin);
}


void generate_random_doubles(double *arr, int n){
    for(int i = 0; i < n; i++){
        arr[i] = fRand(0., 10000.);
    }
}


typedef unsigned long long int ull_t;




__device__ void sleep_cycles(clock_t clock_count)
{
    clock_t start_clock = clock();
    clock_t clock_offset = 0;
    while (clock_offset < clock_count)
    {
        clock_offset = clock() - start_clock;
    }
}


__device__ void mutex_lock(unsigned int *mutex) {
    unsigned int ns = 8;
    while (atomicCAS(mutex, 0, 1) == 1) {
        sleep_cycles(ns);
        if (ns < 256) {
            ns *= 2;
        }
    }
}


__device__ void mutex_unlock(unsigned int *mutex) {
    atomicExch(mutex, 0);
}



__global__ void cuda_reduction(ull_t *arr, int n, ull_t *ret) {
   
    unsigned int tid = threadIdx.x;

    ull_t item_local = arr[tid];
    __shared__ unsigned int mutex[1];
    mutex[0] = 0;

    __syncthreads();

    mutex_lock(mutex);

    if(*ret > item_local){
        *ret = item_local;
    }

    mutex_unlock(mutex);
}




int main() {

    ull_t *ret_ull = new ull_t;
    double *ret_double = new double;
    double *arr = new double[N];
    ull_t *arr_ull = new ull_t[N];
    
    generate_random_doubles(arr, N);

    std::cout << "[main] Generated numbers:";
    for (int i = 0; i < N; i++) {
        std::cout << ' ' << arr[i];
    }
    std::cout << '\n';


    ull_t mask = ((ull_t)1) << 63; 

    memcpy(arr_ull, arr, N * sizeof(ull_t));

    for(int i = 0;i < N; i++){
        arr_ull[i] ^= mask;
    }


    ull_t *ret_ull_dev;
    ull_t *arr_ull_dev;
    hipMalloc(&arr_ull_dev, N * sizeof(ull_t));
    hipMalloc(&ret_ull_dev, 1 * sizeof(ull_t));
    hipMemcpy((BYTE *)arr_ull_dev, (BYTE *)arr_ull, N * sizeof(ull_t), hipMemcpyHostToDevice);
    hipMemcpy((BYTE *)ret_ull_dev, (BYTE *)arr_ull, 1 * sizeof(ull_t), hipMemcpyHostToDevice);
                                   
    cuda_reduction<<<1, N>>>(arr_ull_dev, N, ret_ull_dev);

    hipDeviceSynchronize();

    hipMemcpy((BYTE *)ret_ull, (BYTE *)ret_ull_dev, 1 * sizeof(ull_t), hipMemcpyDeviceToHost);
    
    (*ret_ull) ^= mask;
    memcpy(ret_double, ret_ull, sizeof(ull_t));
    std::cout << "[main] (cuda) The minimum value: " << *ret_double << '\n';

    *ret_double = CPU_reduction(arr, N);
    std::cout << "[main] (cpu) The minimum value: " << *ret_double << '\n';
    
    delete ret_ull;
    delete ret_double;
    delete [] arr;
    return 0;
}
